#include "hip/hip_runtime.h"
#include <seuimage/seuimage.hpp>

#define BLOCKX 16
#define BLOCKY 16

__device__ unsigned char rgb_bound(int v)
{
  return v > 255 ? 255 : (v < 0 ? 0 : v);
}

__global__ void baygr2bgr_kernal(uint8_t *bayergr, uint8_t *bgr, int w, int h, 
    float rgain, float ggain, float bgain)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int outy = h - y;
  int outx = w - x;
  float r, g, b;

  b = bayergr[(y + ((y + 1) & 1)) * w + x - (x & 1)] * bgain;
  g = bayergr[y * w + x - (x & 1) + (y & 1)] * ggain;
  r = bayergr[(y - (y & 1)) * w + x + ((x + 1) & 1)] * rgain;

  bgr[outy * w * 3 + outx * 3 + 0] = rgb_bound(r);
  bgr[outy * w * 3 + outx * 3 + 1] = rgb_bound(g);
  bgr[outy * w * 3 + outx * 3 + 2] = rgb_bound(b);
}

__global__ void white_balance_kernal(unsigned char *rgb, float rgain, float ggain, float bgain)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int w = gridDim.x * blockDim.x;

  int offset = y * w * 3 + x * 3;
  float rf, gf, bf;
  rf = rgb[offset + 0] * rgain;
  gf = rgb[offset + 1] * ggain;
  bf = rgb[offset + 2] * bgain;
  rgb[offset + 0] = rgb_bound(rf);
  rgb[offset + 1] = rgb_bound(gf);
  rgb[offset + 2] = rgb_bound(bf);
}

__global__ void label_color_kernal(unsigned char *rgb, unsigned char *hsv, int h_l, int h_h, int s_l, int s_h, int v_l,
                                   int v_h)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int w = gridDim.x * blockDim.x;

  int offset = y * w * 3 + x * 3;
  unsigned char h = hsv[offset + 0];
  unsigned char s = hsv[offset + 1];
  unsigned char v = hsv[offset + 2];
  if (h >= h_l && h <= h_h && s >= s_l && s <= s_h && v >= v_l && v <= v_h)
  {
    rgb[offset + 0] = 0;
    rgb[offset + 1] = 255;
    rgb[offset + 2] = 0;
  }
}

__global__ void build_map_kernal(float *pCamK, float *pDistort, float *pInvNewCamK, float *pMapx, float *pMapy,
                                 int outImgW, int outImgH)
{
  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
  if (tidx < outImgW && tidy < outImgH)
  {
    float k1 = pDistort[0];
    float k2 = pDistort[1];
    float p1 = pDistort[2];
    float p2 = pDistort[3];
    float k3, k4, k5, k6, s1, s2, s3, s4;
    k3 = k4 = k5 = k6 = s1 = s2 = s3 = s4 = 0;
    float fx = pCamK[0];
    float fy = pCamK[4];
    float u0 = pCamK[2];
    float v0 = pCamK[5];

    float _x = tidx * pInvNewCamK[0] + tidy * pInvNewCamK[1] + pInvNewCamK[2];
    float _y = tidx * pInvNewCamK[3] + tidy * pInvNewCamK[4] + pInvNewCamK[5];
    float _w = tidx * pInvNewCamK[6] + tidy * pInvNewCamK[7] + pInvNewCamK[8];

    float w = 1. / _w;
    float x = _x * w;
    float y = _y * w;

    float x2 = x * x;
    float y2 = y * y;
    float r2 = x2 + y2;
    float _2xy = 2 * x * y;
    float kr = (1 + ((k3 * r2 + k2) * r2 + k1) * r2) / (1 + ((k6 * r2 + k5) * r2 + k4) * r2);
    float xd = (x * kr + p1 * _2xy + p2 * (r2 + 2 * x2) + s1 * r2 + s2 * r2 * r2);
    float yd = (y * kr + p1 * (r2 + 2 * y2) + p2 * _2xy + s3 * r2 + s4 * r2 * r2);

    float invProj = 1.;
    float u = fx * invProj * xd + u0;
    float v = fy * invProj * yd + v0;

    int mapIdx = tidy * outImgW + tidx;
    pMapx[mapIdx] = (float)u;
    pMapy[mapIdx] = (float)v;
  }
}

__global__ void remap_kernal(unsigned char *pSrcImg, unsigned char *pDstImg, float *pMapx, float *pMapy, int inWidth,
                             int inHeight, int outWidth, int outHeight, int channels)
{
  const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  const int tidy = blockDim.y * blockIdx.y + threadIdx.y;
  if (tidx < outWidth && tidy < outHeight)
  {
    int mapIdx = tidy * outWidth + tidx;
    float u = pMapx[mapIdx];
    float v = pMapy[mapIdx];

    int u1 = floor(u);
    int v1 = floor(v);
    int u2 = u1 + 1;
    int v2 = v1 + 1;
    if (u1 >= 0 && v1 >= 0 && u2 < inWidth && v2 < inHeight)
    {
      float dx = u - u1;
      float dy = v - v1;
      float weight1 = (1 - dx) * (1 - dy);
      float weight2 = dx * (1 - dy);
      float weight3 = (1 - dx) * dy;
      float weight4 = dx * dy;

      int resultIdx = mapIdx * 3;
      for (int chan = 0; chan < channels; chan++)
      {
        pDstImg[resultIdx + chan] = (unsigned char)(weight1 * pSrcImg[(v1 * inWidth + u1) * 3 + chan] +
                                                    weight2 * pSrcImg[(v1 * inWidth + u2) * 3 + chan] +
                                                    weight3 * pSrcImg[(v2 * inWidth + u1) * 3 + chan] +
                                                    weight4 * pSrcImg[(v2 * inWidth + u2) * 3 + chan]);
      }
    }
  }
}

namespace seuimage
{
bool WhiteBalance(CudaMatC &rgb, float rgain, float ggain, float bgain)
{
  int w = rgb.width(), h = rgb.height();
  dim3 block(BLOCKX, BLOCKY);
  dim3 grid(w / BLOCKX, h / BLOCKY);
  white_balance_kernal<<<grid, block>>>(rgb.data(), rgain, ggain, bgain);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return false;
  return true;
}

bool Undistored(CudaMatC &in, CudaMatC &out, float *pCamK, float *pDistort, float *pInvNewCamK, float *pMapx,
                float *pMapy)
{
  int w = in.width(), h = in.height(), c = in.channels();
  dim3 block(BLOCKX, BLOCKY);
  dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
  build_map_kernal<<<grid, block>>>(pCamK, pDistort, pInvNewCamK, pMapx, pMapy, w, h);
  hipDeviceSynchronize();
  remap_kernal<<<grid, block>>>(in.data(), out.data(), pMapx, pMapy, w, h, w, h, c);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return false;
  return true;
}

bool LabelColor(CudaMatC &rgb, CudaMatC &hsv, const std::vector<cv::Point> &HSV)
{
  int h_l = HSV[0].x, h_h = HSV[0].y;
  int s_l = HSV[1].x, s_h = HSV[1].y;
  int v_l = HSV[2].x, v_h = HSV[2].y;

  int w = rgb.width(), h = rgb.height();
  dim3 block(BLOCKX, BLOCKY);
  dim3 grid(w / BLOCKX, h / BLOCKY);
  label_color_kernal<<<grid, block>>>(rgb.data(), hsv.data(), h_l, h_h, s_l, s_h, v_l, v_h);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    return false;
  return true;
}

bool CudaBayerToRGB(CudaMatC &bayer, CudaMatC &rgb, float rgain, float ggain, float bgain)
{
    int wi = bayer.width(), hi = bayer.height();
    int wo = rgb.width(), ho = bayer.height();
    if(wi!=wo || hi!=ho) return false;
    dim3 block(BLOCKX, BLOCKY);
    dim3 grid(wi / BLOCKX, hi / BLOCKY);
    baygr2bgr_kernal<<<grid, block>>>(bayer.data(), rgb.data(), wi, hi, rgain, ggain, bgain);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
        return false;
    return true;
}
}  // namespace seuimage